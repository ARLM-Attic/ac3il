#include "hip/hip_runtime.h"
﻿#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include <cuPrintf.cu>

//The macro CUPRINTF is defined for architectures
//with different compute capabilities.
#if __CUDA_ARCH__ < 200 	//Compute capability 1.x architectures
#define CUPRINTF cuPrintf
#else						//Compute capability 2.x architectures
#define CUPRINTF(fmt, ...) printf("[%d, %d]:\t" fmt, \
								blockIdx.y*gridDim.x+blockIdx.x,\
								threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
								__VA_ARGS__)
#endif

extern "C" __global__ void VecAdd(float* A, float* B, float* C){
	int i = threadIdx.x;
	C[i] = A[i] + B[i];

	CUPRINTF("Computed value is:%d\n", C[0]);
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_